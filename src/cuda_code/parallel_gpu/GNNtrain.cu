#include "hip/hip_runtime.h"
#include "GNNConfig.h"
#include "../common/Time.h"

/*
 * @author Vasileios Zois
 * @email vzois@usc.edu
 *
 * CUDA neural network implementation
 */

#define MBATCH 128
#define TTILE 32
#define LTILE 32

#define DPT 4 //DATA PER THREADS
#define BSIZE 512

namespace gnn_kernels{
	static __device__ hiprandState randDevStates[RAND_STATES];

	__device__ float cudaUniRand(unsigned int tid){
		return hiprand_uniform(&randDevStates[tid % RAND_STATES]);
	}

	__global__ void cudaSetupRandStatesKernel(unsigned int seed){
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	hiprand_init(seed, blockIdx.x, 0, &randDevStates[i]);
	}

	__host__ void cudaInitRandStates(){
		dim3 grid = grid_1D(RAND_STATES,RAND_BLOCK_THREADS);
		dim3 block = block_1D(RAND_BLOCK_THREADS);

		Utils<unsigned int> u;
		cudaSetupRandStatesKernel<<<grid,block>>>(u.uni(UINT_MAX));
		handleDeviceErrors(hipDeviceSynchronize(),"Error initializing random states");
	}


	/*
	 * Testing activation functions on kernels.
	 */
	template<typename ACT_F>
	__global__ void bench_test_activation(ACT_F F){
		int i = threadIdx.x + blockIdx.x * blockDim.x;
		float a = 0;
		for(int j = 0; j<blockDim.x;j++){
			a+= F.F((float)i*j);
		}
	}

	/*
	 * Initialize matrices random weights
	 */
	template<typename DATA_T>
	__global__ void randomWeights(DATA_T *W_j,unsigned int rows, unsigned int cols){
		int i = blockIdx.x * blockDim.x + threadIdx.x;

		if( i < rows * cols){
			if((i+1) % cols == 0){
				W_j[i] = 0.0;
			}else{
				W_j[i] = (2.0 * cudaUniRand(i) - 1.0) * (sqrtf(6.0)/ (rows + cols));
			}
		}
	}
	/*
	 * Load current batch of train examples.
	 * 		1: First layer batch array.
	 * 		2: Training example matrix.
	 * 		3: Input layer dimension
	 * 		4: Batch size dimension
	 * 		5: Offset indicating the batch being loaded.
	 * 	Notes:
	 * 		Transpose version assumes that the training examples matrix is stored
	 * 		in a row-wise manner.
	 */
	template<typename DATA_T, unsigned int TILE>
	__global__ void loadT(
			DATA_T *A_j,
			DATA_T *tEx,
			unsigned int clayer, unsigned int bsize,
			unsigned int car, unsigned int dim,
			unsigned int voffset, unsigned int hoffset
			){

		__shared__ DATA_T stEx[TILE * TILE];
		int row = ( blockIdx.y * blockDim.y + threadIdx.y );
		int col = ( blockIdx.x * blockDim.x + threadIdx.x );

		if(voffset + row < car && col + hoffset < dim && row < bsize && col < clayer){
			stEx[threadIdx.y * TILE + threadIdx.x] = tEx[(row + voffset) * dim + (col + hoffset)];
			//stEx[threadIdx.y * TILE + threadIdx.x] = tEx[(voffset + row) * dim + (col + hoffset)];
		}
		__syncthreads();

		//col * bsize + row
		row = (blockIdx.x * blockDim.x + threadIdx.y);
		col = (blockIdx.y * blockDim.y + threadIdx.x);
		if( row < clayer && col < bsize){
			A_j[row * bsize + col] = stEx[threadIdx.x * TILE + threadIdx.y];
		}
	}

	/*
	 * Compute matrix of activation values for a single layer of a given batch.
	 *		1:	Current layer weight matrix.
	 *		2: 	Current layer matrix of activation vectors.
	 *		3: 	Next layer matrix of activation vectors.
	 *		4:	W_j = nlayer x clayer , A_j = clayer x bsize, A_jj = nlayer x bsize.
	 *		5: 	Offset: 0 for hidden and output layer, corresponding row of training example matrix for input layer.
	 */
	template<typename DATA_T, typename ACT_F, unsigned int TILE>
	__global__ void	mmul(
			DATA_T *A_jj,
			DATA_T *W_j,
			DATA_T *A_j,
			ACT_F F,
			unsigned int nlayer,
			unsigned int clayer,
			unsigned int bsize
			)
	{
		__shared__ DATA_T sWj[TILE * TILE];
		__shared__ DATA_T sAj[TILE * TILE];
		__shared__ DATA_T bias[TILE];

		int row = ( blockIdx.y * blockDim.y + threadIdx.y );
		int col = ( blockIdx.x * blockDim.x + threadIdx.x );
		if(threadIdx.x == 0) bias[threadIdx.y] = W_j[row * (clayer + 1) + clayer];
		__syncthreads();
		DATA_T Ajj = bias[threadIdx.y];

		int loadOffset = threadIdx.y*TILE + threadIdx.x;
		for(int i = 0;i < ((clayer - 1) / TILE) + 1; i++){
			if( row < nlayer && (i * TILE + threadIdx.x ) < clayer)
				sWj[loadOffset] = W_j[ row * ( clayer + 1 ) + i * TILE  + threadIdx.x];// clayer + 1  to avoid bias vector
			else sWj[loadOffset] = 0.0;

			if ( i*TILE + threadIdx.y < clayer && col < bsize )
				sAj[loadOffset] = A_j[(i * TILE + threadIdx.y) * bsize + col];
			else sAj[loadOffset] = 0.0;
			__syncthreads();

			for(int j = 0;j < TILE; j++){
				Ajj += sWj[threadIdx.y * TILE + j] * sAj[j * TILE + threadIdx.x];
			}

			__syncthreads();
		}

		if( row < nlayer && col < bsize )
			A_jj[row * bsize + col ] = F.F(Ajj);
	}

	/*
	 * Kernel that computes the last layer difference between the batch activation matrix and the expected output
	 * matrix.
	 */
	template<typename DATA_T>
	__global__ void outputD(
			DATA_T *D_j,
			DATA_T *ExA_j,
			DATA_T *A_j,
			unsigned int size
		)
	{
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		if ( i < size){ D_j[i] = ExA_j[i] - A_j[i]; }
	}

	/*
	 * Transpose matrix multiplication.
	 *  D_j = (W_j)^T . D_jj
	 */
	template<typename DATA_T, unsigned int TILE>
	__global__ void tmmul(
			DATA_T *D_j,
			DATA_T *W_j,
			DATA_T *D_jj,
			unsigned int clayer,
			unsigned int nlayer,
			unsigned int bsize
			)
	{
		__shared__ DATA_T sWj[TILE * TILE];
		__shared__ DATA_T sDjj[TILE * TILE];

		DATA_T Dj = 0.0;
		int colW = ( blockIdx.y * blockDim.y + threadIdx.x );// by * TILE + ty * clayer + threadIdx.x
		int colD = ( blockIdx.x * blockDim.x + threadIdx.x );

		int loadOffset = threadIdx.y*TILE + threadIdx.x;
		for(int i = 0; i < (nlayer - 1) / TILE + 1 ; i++){
			if( (i * TILE +  threadIdx.y) < nlayer && colW < clayer)
				sWj[loadOffset] = W_j[ (i * TILE +  threadIdx.y) * clayer + colW ];
			else
				sWj[loadOffset] = 0.0;

			if((i * TILE + threadIdx.y) < nlayer && colD < bsize)
				sDjj[loadOffset] = D_jj[ (i * TILE + threadIdx.y) * bsize + colD ];
			else
				sDjj[loadOffset] = 0.0;
			__syncthreads();

			for(int j=0;j<TILE;j++) Dj += sWj[j * TILE + threadIdx.y] * sDjj[j * TILE + threadIdx.x];
			__syncthreads();
		}

		int row = ( blockIdx.y * blockDim.y + threadIdx.y );
		if( row < clayer && colD < bsize) D_j[row * bsize + colD] = Dj;
	}

	/*
	 *	Hadamard product with derivative activation values
	 */
	template<typename DATA_T, typename ACT_F, unsigned int TILE>
	__global__ void	hmprod_mmul(
			DATA_T *D_j,
			DATA_T *W_j,
			DATA_T *A_j,
			ACT_F F,
			unsigned int nlayer,
			unsigned int clayer,
			unsigned int bsize
			)
	{
		__shared__ DATA_T sWj[TILE * TILE];
		__shared__ DATA_T sAj[TILE * TILE];
		__shared__ DATA_T bias[TILE];

		int row = ( blockIdx.y * blockDim.y + threadIdx.y );
		int col = ( blockIdx.x * blockDim.x + threadIdx.x );
		if(threadIdx.x == 0) bias[threadIdx.y] = W_j[row * (clayer + 1) + clayer];
		__syncthreads();
		DATA_T Dj = bias[threadIdx.y];

		int loadOffset = threadIdx.y*TILE + threadIdx.x;
		for(int i = 0;i < ((clayer - 1) / TILE) + 1; i++){
			if( row < nlayer && (i * TILE + threadIdx.x ) < clayer)
				sWj[loadOffset] = W_j[ row * ( clayer + 1 ) + i * TILE  + threadIdx.x];// clayer + 1  to avoid bias vector
			else sWj[loadOffset] = 0.0;

			if ( i*TILE + threadIdx.y < clayer && col < bsize )
				sAj[loadOffset] = A_j[(i * TILE + threadIdx.y) * bsize + col];
			else sAj[loadOffset] = 0.0;
			__syncthreads();

			for(int j = 0;j < TILE; j++) Dj += sWj[threadIdx.y * TILE + j] * sAj[j * TILE + threadIdx.x];
			__syncthreads();
		}


		if( row < nlayer && col < bsize )
			D_j[row * bsize + col ] *= F.D(Dj);
	}

	template<typename DATA_T, typename ACT_F, unsigned int TILE>
	__global__ void	hmprod_mmul_sigmoid(
			DATA_T *D_j,
			DATA_T *A_j,
			unsigned int nlayer,
			unsigned int clayer,
			unsigned int bsize
			)
	{

		int row = ( blockIdx.y * blockDim.y + threadIdx.y );
		int col = ( blockIdx.x * blockDim.x + threadIdx.x );
		DATA_T Aj = A_j[row * bsize + col];

		if( row < nlayer && col < bsize )
			D_j[row * bsize + col ] *= Aj * (1-Aj);
	}


	/*
	 * 	Compute weight update matrices for the current batch.
	 * 	A = [ A ones(bsize) ]
	 *	for i = 1 : dsz(2)
	 *		W = W + D(:,i) * A(:,i)';
	 *	end
	 *	W ( nlayer x (clayer + 1))
	 */
	template<typename DATA_T, unsigned int TILE>
	__global__ void tvecpvec(
			DATA_T *W_j,
			DATA_T *D_jj,
			DATA_T *A_j,
			unsigned int nlayer,
			unsigned int bsize,
			unsigned int clayer,
			float lrate
			){

		__shared__ DATA_T sDjj[TILE * TILE];
		__shared__ DATA_T sAj[TILE * TILE];

		DATA_T Wj = 0.0;
		int rowD = (blockIdx.y * blockDim.y + threadIdx.y);
		int rowA = (blockIdx.x * blockDim.x + threadIdx.y);

		for(int i = 0;i < (bsize - 1) / TILE + 1;i++){
			if(rowD < nlayer && (i*TILE + threadIdx.x) < bsize)
				sDjj[threadIdx.y * TILE + threadIdx.x] = D_jj[rowD * bsize + i*TILE + threadIdx.x];
			else
				sDjj[threadIdx.y * TILE + threadIdx.x] = 0.0;

			if(rowA < clayer && (i*TILE + threadIdx.x) < bsize)
				sAj[threadIdx.x * TILE + threadIdx.y] = A_j[rowA * bsize + i*TILE + threadIdx.x];
			else
				sAj[threadIdx.x * TILE + threadIdx.y] = 1.0;//Required to update bias weights//
			__syncthreads();

			for(int j = 0 ; j < TILE; j++)
				Wj += sDjj[threadIdx.y * TILE + j] * sAj[j * TILE + threadIdx.x];
			__syncthreads();
		}

		int col = (blockIdx.x * blockDim.x + threadIdx.x);
		Wj *= (lrate / bsize);
		if( rowD < nlayer && col < clayer + 1)//clayer + 1 to update bias weights.
			W_j[rowD * (clayer + 1) + col] += Wj;
	}

	template<typename DATA_T,unsigned int init>
	__global__ void initVector(DATA_T *M, unsigned int rows, unsigned int cols){
		int i = threadIdx.x + blockDim.x * blockIdx.x;

		while( i < rows * cols){
			if (init == ZEROS ) M[i] = 0.0;
			else if (init == ONES) M[i] = 1.0;
			else if (init == RANDOM) M[i] = cudaUniRand(i);
			i+=gridDim.x * blockDim.x;
		}
	}


	template<typename DATA_T>
	__global__ void printGPU2(DATA_T *A, unsigned int row, unsigned int col){
		for(int i =0;i<row;i++){
			printf("[ ");
			for(int j=0;j<col;j++){
				printf("%.1f ", A[i*col + j]);
			}
			printf(" ]\n");
		}
	}

	template<typename DATA_T>
	__host__ void printGPU(DATA_T *A, unsigned int row, unsigned int col){
		DATA_T *AA;
		allocHostMem<DATA_T>(&AA,sizeof(DATA_T)*row*col,"Error allocating AA in printGPU");
		safeCpyToHost<DATA_T>(AA,A,sizeof(DATA_T)*row*col,"Error copying AA in printGPU");
		printf("[ ");
		for(int i =0;i<row;i++){
			for(int j=0;j<col;j++){
				printf("%.1f ", AA[i*col + j]);
			}
			printf(" ;\n");
		}
		printf(" ]\n");
	}
}

namespace gnn{

	template<typename DATA_T, typename ACT_F>
	void GNeuralNetwork<DATA_T,ACT_F>::randomInit(){
		if(network == NULL) vz::error("Network architecture missing. Use createLayers first!");
		//std::cout<<"Initializing random weights: "<<std::endl;
		hipSetDevice(CUDA_DEVICE);

		gnn_kernels::cudaInitRandStates();
		for(int i = 0;i < layers-1;i++){
			//std::cout<<network[i].clayer << "{}" << network[i].nlayer << std::endl;
			unsigned int vector_size = network[i].nlayer * network[i].clayer;
			dim3 grid = grid_1D(vector_size,BSIZE);
			dim3 block = block_1D(BSIZE);
			gnn_kernels::randomWeights<DATA_T><<<grid,block>>>(network[i].W_j,network[i].nlayer,network[i].clayer);
			handleDeviceErrors(hipDeviceSynchronize(),"Error executing randomWeights kernel");
		}
	}

	template<typename DATA_T, typename ACT_F>
	void GNeuralNetwork<DATA_T,ACT_F>::train(){
		hipSetDevice(CUDA_DEVICE);
		if(network == NULL) vz::error("Network architecture missing. Use createLayers first!");
		if(bsize == 0) vz::error("Batch size not set. Use setBatchSize first!");
		unsigned int nbatch = this->transpose ? dimEx.first / this->bsize : dimEx.second / this->bsize;

		for(int i = 0; i< nbatch; i++){
			/*
			 * Load current batch of training examples.
			 */
			unsigned int bRow = i * this->bsize;
			dim3 lgrid((batch[0].clayer-1)/LTILE + 1, (batch[0].bsize-1)/LTILE + 1);
			dim3 lblock(LTILE,LTILE);
			gnn_kernels::loadT<DATA_T,LTILE><<<lgrid,lblock>>>(
					batch[0].A_j,dExamples,
					batch[0].clayer,batch[0].bsize,
					dimEx.first,dimEx.second,
					bRow,0
				);
			handleDeviceErrors(hipDeviceSynchronize(),"Error executing loadT for X batch");

			if(DEBUG_T){
			//printf("A=");
			gnn_kernels::printGPU(batch[0].A_j,batch[0].clayer,batch[0].bsize);
			hipDeviceSynchronize();
			printf("sum(sum(round(A - M(%d:%d,1:%d)')))\n",bRow + 1, (i+1)*batch[0].bsize,batch[0].clayer);
			}


			/*
			 * Neural network feed forward step.
			 * 		- W = ( nlayer x (clayer + 1) ), A(i) = ( clayer x bsize ) , A(i+1) = (nlayer x bsize)
			 * 		A[jj] = A[j] * W[j]
			 */
			for(int j = 0;j < this->layers - 1;j++){
				dim3 agrid((batch[j+1].bsize - 1)/TTILE + 1, (batch[j+1].clayer - 1)/TTILE + 1);
				dim3 ablock(TTILE,TTILE);
				gnn_kernels::mmul<DATA_T,ACT_F,TTILE><<<agrid,ablock>>>
						(
								batch[j+1].A_j,
								network[j].W_j,
								batch[j].A_j,
								F,
								network[j].nlayer,
								network[j].clayer - 1,// Ignore bias vector from the multiplication//
								batch[j].bsize
						);
				handleDeviceErrors(hipDeviceSynchronize(),"Error executing batch activation");

				if(DEBUG_GNN){
				printf("Ajj= ");
				gnn_kernels::printGPU(batch[j+1].A_j,batch[j+1].clayer,batch[j+1].bsize);
				hipDeviceSynchronize(); //printf("------------------>\n");
				printf(";W= ");
				gnn_kernels::printGPU(network[j].W_j,network[j].nlayer,network[j].clayer);
				hipDeviceSynchronize(); //printf("------------------>\n");
				printf(";Aj= ");
				gnn_kernels::printGPU(batch[j].A_j,batch[j].clayer,batch[j].bsize);
				hipDeviceSynchronize();
				printf("Ejj=act(W,Aj,0);\n");
				printf("diff0%d = sum(sum(round(Ejj-Ajj)))\n",j);
				}
			}

			/*
			 * Output layer Delta computation.
			 *	Dl = (Y - Al)
			 *
			 */
			dim3 ogrid = grid_1D(batch[layers-1].clayer * batch[layers-1].bsize, BSIZE);
			dim3 oblock = block_1D(BSIZE);

			bRow = i * this->bsize;
			dim3 lygrid((batch[layers-1].clayer-1)/LTILE + 1, (batch[layers-1].bsize-1)/LTILE + 1);
			dim3 lyblock(LTILE,LTILE);
			gnn_kernels::loadT<DATA_T,LTILE><<<lygrid,lyblock>>>(
					batch[layers-1].Y,dExamples,
					batch[layers-1].clayer,batch[layers-1].bsize,
					dimEx.first,dimEx.second,
					bRow,batch[0].clayer
			);

			if(DEBUG_T){
			handleDeviceErrors(hipDeviceSynchronize(),"Error executing loadT for Y batch");
			printf("Y=");
			gnn_kernels::printGPU(batch[layers-1].Y,batch[layers-1].clayer,batch[layers-1].bsize);
			hipDeviceSynchronize();
			printf("sum(sum(round(Y - M(%d:%d,%d:%d)')))\n",bRow + 1, (i+1)*batch[0].bsize,batch[0].clayer+1,batch[0].clayer+batch[layers-1].clayer);
			}

			gnn_kernels::outputD<DATA_T><<<ogrid,oblock>>>(
					batch[layers-1].D_j,
					batch[layers-1].Y,
					batch[layers-1].A_j,// Dj =  Y - Aj
					batch[layers-1].clayer * batch[layers-1].bsize
				);
			handleDeviceErrors(hipDeviceSynchronize(),"Error executing outputD kernel");

			if(DEBUG_GNN){
			printf("Y=");
			gnn_kernels::printGPU(batch[layers-1].Y,batch[layers-1].clayer,batch[layers-1].bsize);
			hipDeviceSynchronize();
			//if(DEBUG_GNN){
			printf(";Aj=");
			gnn_kernels::printGPU(batch[layers-1].A_j,batch[layers-1].clayer,batch[layers-1].bsize);
			hipDeviceSynchronize();
			printf(";Dl=");
			gnn_kernels::printGPU(batch[layers-1].D_j,batch[layers-1].clayer,batch[layers-1].bsize);
			hipDeviceSynchronize();
			printf(";El = (Y - Aj);\n");
			printf("diffY = sum(sum(round(El - Dl)))\n");
			}

			/*
			 * Backpropagation transpose matrix multiplication.
			 * 		for i = layers-1 : > 1 : i--
			 * 		batch[i-1].D_j = network[i-1].W_j * batch[i].D_j
			 * 		grid = (batch[i-1].bsize / TILE + 1), batch[i-1].clayer / TILE + 1
			 * 		block = (TILE, TILE)
			 * 		D[j] = <W[j] * D[jj]> .* F.D(W[j] * A[j])
			 */
			for(int j = layers-1; j > 1 ; j--){
					//printf("BP(%d)\n",j);
					dim3 dgrid((batch[j-1].bsize - 1) / TTILE + 1, (batch[j-1].clayer - 1) / TTILE + 1);
					dim3 dblock(TTILE, TTILE);
					gnn_kernels::tmmul<DATA_T,TTILE><<<dgrid,dblock>>>(
							batch[j-1].D_j,//(clayer x bsize)
							network[j-1].W_j,//(nlayer x clayer)
							batch[j].D_j,// (nlayer x bsize)
							network[j-1].clayer,
							network[j-1].nlayer,
							batch[j].bsize
							);
					handleDeviceErrors(hipDeviceSynchronize(),"Error executing tmmul kernel");
					if(DEBUG_GNN){
					printf("Djj=");
					gnn_kernels::printGPU(batch[j-1].D_j,batch[j-1].clayer,batch[j-1].bsize);
					hipDeviceSynchronize(); //printf("------------------>\n");
					printf(";W=");
					gnn_kernels::printGPU(network[j-1].W_j,network[j-1].nlayer,network[j-1].clayer);
					hipDeviceSynchronize();//printf("------------------>\n");
					printf(";Dj=");
					gnn_kernels::printGPU(batch[j].D_j,batch[j].clayer,batch[j].bsize);
					hipDeviceSynchronize();//printf("------------------>\n");
					printf("Ejj=W' * Dj; \n");
					printf("diff1%d = sum(sum(round(Ejj(1:%d,:) - Djj)))\n",j-1,network[j-1].clayer-1);
					}
			}

			/*
			 * Final step for delta computation.
			 * 		//D[jj] = D[jj] .* F.D(W[j] * A(j))
			 */
			for(int j = 1; j < layers-1; j++){
				dim3 dgrid((batch[j].bsize - 1) / TTILE + 1, (batch[j].clayer - 1) / TTILE + 1);
				dim3 dblock(TTILE, TTILE);

				if(DEBUG_GNN){
					printf("Djj=");
					gnn_kernels::printGPU(batch[j].D_j,batch[j].clayer,batch[j].bsize);
					hipDeviceSynchronize(); //printf("------------------>\n");
					printf(";Wj=");
					gnn_kernels::printGPU(network[j-1].W_j,network[j-1].nlayer,network[j-1].clayer);
					hipDeviceSynchronize();//printf("------------------>\n");
					printf(";Aj=");
					gnn_kernels::printGPU(batch[j-1].A_j,batch[j-1].clayer,batch[j-1].bsize);
					hipDeviceSynchronize();//printf("------------------>\n");
				}
				if(strcmp(F.TAG,"Sigmoid")!=0){
					gnn_kernels::hmprod_mmul<DATA_T,ACT_F,TTILE><<<dgrid, dblock>>>(
							batch[j].D_j,
							network[j-1].W_j,
							batch[j-1].A_j,
							F,
							network[j-1].nlayer,
							network[j-1].clayer-1,
							batch[j-1].bsize
							);
				}else{
					gnn_kernels::hmprod_mmul_sigmoid<DATA_T,ACT_F,TTILE><<<dgrid, dblock>>>(
							batch[j].D_j,
							batch[j].A_j,
							network[j-1].nlayer,
							network[j-1].clayer-1,
							batch[j-1].bsize
							);
				}
				handleDeviceErrors(hipDeviceSynchronize(),"Error executing tmmul kernel");//TODO not necessary

				if(DEBUG_GNN){
				printf(";Ejj=");
				gnn_kernels::printGPU(batch[j].D_j,batch[j].clayer,batch[j].bsize);
				hipDeviceSynchronize(); //printf("------------------>\n");
				printf(";Djj = Djj .* act(Wj,Aj,1);\n");
				printf("diff2%d=sum(sum(round(Ejj - Djj)))\n",j);
				}
			}

			/*
			 * Weight and bias update
			 * W[j] = W[j] + (lrate/bsize) * Sum( D[jj] <> A[j] )
			 */
			for(int j = 0;j<layers-1; j++){
				dim3 grid((network[j].clayer - 1)/TTILE + 1, (network[j].nlayer - 1)/TTILE + 1 );
				dim3 block(TTILE,TTILE);
				if(DEBUG_GNN){
					printf("Wj=");
					gnn_kernels::printGPU(network[j].W_j,network[j].nlayer,network[j].clayer);
					hipDeviceSynchronize();
					printf(";Djj=");
					gnn_kernels::printGPU(batch[j+1].D_j,batch[j+1].clayer,batch[j+1].bsize);
					hipDeviceSynchronize();
					printf(";Aj=");
					gnn_kernels::printGPU(batch[j].A_j,batch[j].clayer,batch[j].bsize);
					hipDeviceSynchronize();
				}

				gnn_kernels::tvecpvec<DATA_T,TTILE><<<grid,block>>>(
					network[j].W_j,
					batch[j+1].D_j,
					batch[j].A_j,
					network[j].nlayer,
					batch[j].bsize,
					network[j].clayer-1,
					this->lrate
					);
				handleDeviceErrors(hipDeviceSynchronize(),"Error executing tvecpvec kernel");//TODO not needed to wait
				if(DEBUG_GNN){
					printf("Ej=");
					gnn_kernels::printGPU(network[j].W_j,network[j].nlayer,network[j].clayer);
					hipDeviceSynchronize();
					printf("diff3%d=sum(sum(round(Ej-tvecpvec(Wj,Djj,Aj,%f,%d))))\n",j,this->lrate,batch[j].bsize);
				}
			}
		}

	}

	template<typename DATA_T, typename ACT_F>
	void GNeuralNetwork<DATA_T,ACT_F>::classify(){
		hipSetDevice(CUDA_DEVICE);
		if(network == NULL) vz::error("Network architecture missing. Use createLayers first!");
		if(bsize == 0) vz::error("Batch size not set. Use setBatchSize first!");
		unsigned int nbatch = this->transpose ? dimT.first / this->bsize : dimT.second / this->bsize;
		unsigned int count =0;
		if(DEBUG_GNN) std::cout<<dimT.first << "," << dimT.second << std::endl;
		if(DEBUG_GNN) std::cout<< "nbatch: " << nbatch << std::endl;

		DATA_T *Y,*A;
		allocHostMem<DATA_T>(&Y,sizeof(DATA_T)*this->bsize*batch[layers-1].clayer,"Error allocating mem for Y in classify");
		allocHostMem<DATA_T>(&A,sizeof(DATA_T)*this->bsize*batch[layers-1].clayer,"Error allocating mem for A in classify");
		for(int i = 0; i< nbatch; i++){
			/*
			 * Load current batch of training examples.
			 */
			if(this->transpose){
				unsigned int bRow = i * this->bsize;
				dim3 lgrid((batch[0].clayer-1)/TTILE + 1, (batch[0].bsize-1)/TTILE + 1);
				dim3 lblock(TTILE,TTILE);
				//print_grid(lgrid,lblock);
				gnn_kernels::loadT<DATA_T,TTILE><<<lgrid,lblock>>>(
						batch[0].A_j,dTest,
						batch[0].clayer,batch[0].bsize,
						dimT.first,dimT.second,
						bRow,0);
				handleDeviceErrors(hipDeviceSynchronize(),"Error executing loadT for X batch on classify");
			}

			/*
			 * Neural network feed forward step.
			 * 		- W = ( nlayer x (clayer + 1) ), A(i) = ( clayer x bsize ) , A(i+1) = (nlayer x bsize)
			 * 		A[jj] = A[j] * W[j]
			 */
			for(int j = 0;j < this->layers - 1;j++){
				dim3 agrid((batch[j+1].bsize - 1)/TTILE + 1, (batch[j+1].clayer - 1)/TTILE + 1);
				dim3 ablock(TTILE,TTILE);
				gnn_kernels::mmul<DATA_T,ACT_F,TTILE><<<agrid,ablock>>>
						(
							batch[j+1].A_j,
							network[j].W_j,
							batch[j].A_j,
							F,
							network[j].nlayer,
							network[j].clayer - 1,// Ignore bias vector from the multiplication//
							batch[j].bsize
						);
				handleDeviceErrors(hipDeviceSynchronize(),"Error executing batch activation");
			}

			unsigned int bRow = i * this->bsize;
			dim3 lgrid((batch[layers-1].clayer-1)/TTILE + 1, (batch[layers-1].bsize-1)/TTILE + 1);
			dim3 lblock(TTILE,TTILE);
			gnn_kernels::loadT<DATA_T,TTILE><<<lgrid,lblock>>>(
					batch[layers-1].Y,dTest,
					batch[layers-1].clayer,batch[layers-1].bsize,
					dimT.first,dimT.second,
					bRow,batch[0].clayer
			);
			handleDeviceErrors(hipDeviceSynchronize(),"Error executing loadT for Y batch on classify");

			safeCpyToHost<DATA_T>(Y,batch[layers-1].Y,sizeof(DATA_T)*batch[layers-1].clayer*this->bsize,"Error transferring Y from GPU");
			safeCpyToHost<DATA_T>(A,batch[layers-1].A_j,sizeof(DATA_T)*batch[layers-1].clayer*this->bsize,"Error transferring A_j from GPU");
			for(int x=0; x < bsize; x++){
				DATA_T maxY = 0, maxA=0;
				int indexY = 0, indexA=0;
				for(int y = 0; y < batch[layers-1].clayer; y++){
					if(Y[y * bsize + x] > maxY){ maxY = Y[y * bsize + x]; indexY = y;}
					if(A[y * bsize + x] > maxA){ maxA = A[y * bsize + x]; indexA = y;}
				}
				if(indexY == indexA ) count++;
			}
		}
		printf("Accuracy: %2.f, %u, %lu\n",(((float)count)/dimT.first)*100,count, dimT.first);
		hipHostFree(Y); hipHostFree(A);
	}

	/*
	 * Testing methods
	 */
	template<typename DATA_T, typename ACT_F>
	void GNeuralNetwork<DATA_T,ACT_F>::bench_act(){
		hipSetDevice(0);
		dim3 block(512,1,1);
		dim3 grid(128,1,1);

		/*
		 * Warm up device
		 */
		gnn_kernels::bench_test_activation<ACT_F><<<grid,block>>>(this->F);
		hipDeviceSynchronize();
		/* <END> */

		std::string msg("Benchmark ");
		msg.append(F.TAG);
		Time<millis> t;
		t.start();
		gnn_kernels::bench_test_activation<ACT_F><<<grid,block>>>(F);
		hipDeviceSynchronize();
		t.lap(msg);
	}

	template<typename DATA_T, typename ACT_F>
	void GNeuralNetwork<DATA_T,ACT_F>::print_weights(){
		DATA_T *cW_j;
		hipSetDevice(0);

		for(int i = 0;i < layers-1;i++){
			unsigned int size = network[i].nlayer * network[i].clayer;
			allocHostMem<DATA_T>(&cW_j,sizeof(DATA_T)*size, "Error Allocating Host Weight Matrix");
			safeCpyToHost<DATA_T>(cW_j,network[i].W_j,sizeof(DATA_T)*size, "Error Allocating Copying Weight Matrix From Device");

			printf("W%d=[",i);
			for(int j = 0;j<size;j++){
				std::cout<<cW_j[j] << " ";
				if((j+1)%network[i].clayer == 0) std::cout<<std::endl;
			}
			printf("]");
			std::cout<<std::endl;
		}

		for(int i = 0;i < layers-1;i++) printf("A%d=act(W%d,A%d,0)\n",i+1,i,i);

	}

	template<typename DATA_T, typename ACT_F>
	void GNeuralNetwork<DATA_T,ACT_F>::bench_test_kernels(UnitTest test,unsigned int m, unsigned int n, unsigned int k,
			bool debug){
		unsigned int nlayer = m, clayer = n + 1, bsize = k;

		DATA_T *hostA, *hostB, *hostC, *hostD;
		DATA_T *devA, *devB, *devC;

		allocDevMem<DATA_T>(&devA,sizeof(DATA_T) * nlayer * clayer, "Error allocating devA memory");
		allocDevMem<DATA_T>(&devB,sizeof(DATA_T) * clayer * bsize, "Error allocating devB memory");
		allocDevMem<DATA_T>(&devC,sizeof(DATA_T) * nlayer * bsize, "Error allocating devC memory");

		allocHostMem<DATA_T>(&hostA,sizeof(DATA_T) * nlayer * clayer, "Error allocating devA memory");
		allocHostMem<DATA_T>(&hostB,sizeof(DATA_T) * clayer * bsize, "Error allocating devB memory");
		allocHostMem<DATA_T>(&hostC,sizeof(DATA_T) * nlayer * bsize, "Error allocating devC memory");

		dim3 rgrid;
		dim3 rblock = block_1D(256);
		rgrid = grid_1D(nlayer * clayer,256); gnn_kernels::randomWeights<DATA_T><<<rgrid,rblock>>>(devA,nlayer, clayer);
		rgrid = grid_1D(clayer * bsize,256); gnn_kernels::randomWeights<DATA_T><<<rgrid,rblock>>>(devB,clayer, bsize);
		rgrid = grid_1D(nlayer * bsize,256); gnn_kernels::randomWeights<DATA_T><<<rgrid,rblock>>>(devC,nlayer,bsize);

		if(test == MMUL){
			dim3 agrid((bsize - 1)/TTILE + 1, (nlayer - 1)/TTILE + 1);
			dim3 ablock(TTILE,TTILE);
			Time<millis> t;
			t.start();
			gnn_kernels::mmul<DATA_T,ACT_F,TTILE><<<agrid,ablock>>>
					(
							devC,
							devA,
							devB,
							F,
							nlayer,
							clayer - 1,
							bsize
					);
			handleDeviceErrors(hipDeviceSynchronize(),"Error executing batch mmul");
			t.lap("GPU serial mmul elapsed time");

			allocHostMem<DATA_T>(&hostD,sizeof(DATA_T) * nlayer * bsize, "Error allocating devC memory");
			safeCpyToHost<DATA_T>(hostA,devA,sizeof(DATA_T)*nlayer*clayer,"Error copying devA to host");
			safeCpyToHost<DATA_T>(hostB,devB,sizeof(DATA_T)*clayer*bsize,"Error copying devB to host");
			safeCpyToHost<DATA_T>(hostC,devC,sizeof(DATA_T)*nlayer*bsize,"Error copying devC to host");

			/*t.start();
			for(int x = 0; x < nlayer; x++){//3
				for (int y = 0; y < bsize; y++){//3
					hostD[x * bsize + y] = hostA[x * (clayer) + clayer - 1];
					for (int z = 0; z < clayer - 1; z++){//2
						hostD[x * bsize + y] += hostA[x * (clayer) + z] * hostB[z * bsize + y];
					}
					hostD[x * bsize + y] = F.f(hostD[x * bsize + y]);
				}
			}
			if(!debug) t.lap("CPU serial mmul elapsed time");
			if(debug){
				gnn_kernels::printGPU(devC,m,k);
				hipDeviceSynchronize(); printf("<----->\n");
				gnn_kernels::printGPU(devA,m,n);
				hipDeviceSynchronize(); printf("<----->\n");
				gnn_kernels::printGPU(devB,n,k);
				hipDeviceSynchronize(); printf("<----->\n");
				for(int x = 0; x<m * k;x++){
					printf("%.4f ", hostD[x]);
					if((x+1)%k==0) printf("\n");
				}
			}else{
				for(int x = 0; x<nlayer * bsize;x++){
					if(((hostD[x] - hostC[x]) > 0.001 )){
						printf("Result matrices do not match(%f,%f)!!!\n",hostD[x],hostC[x] );
					}
				}
			}
			hipHostFree(hostD);*/
		}else if(test == TMMUL){
			// devB = devA * devC
			// (n x k) = (m x n) (m x k) <=> (n x k) = (m x n)^T (m x k) <=> (n x k) = (n x m) (m x k)
			Time<millis> t;
			dim3 agrid((bsize - 1)/TTILE + 1, (clayer - 1)/TTILE + 1);
			dim3 ablock(TTILE,TTILE);
			t.start();
			gnn_kernels::tmmul<DATA_T,TTILE><<<agrid,ablock>>>(
					devB,//n
					devA,//
					devC,//
					clayer,
					nlayer,
					bsize
			);
			handleDeviceErrors(hipDeviceSynchronize(),"Error executing tmmul kernel");
			t.lap("GPU serial tmmul elapsed time");

			allocHostMem<DATA_T>(&hostD,sizeof(DATA_T) * clayer * bsize, "Error allocating devC memory");
			safeCpyToHost<DATA_T>(hostA,devA,sizeof(DATA_T)*nlayer*clayer,"Error copying devA to host");
			safeCpyToHost<DATA_T>(hostB,devB,sizeof(DATA_T)*clayer*bsize,"Error copying devB to host");
			safeCpyToHost<DATA_T>(hostC,devC,sizeof(DATA_T)*nlayer*bsize,"Error copying devC to host");

			/*t.start();
			for(int x = 0; x < clayer; x++){//3
				for (int y = 0; y < bsize; y++){//3
					hostD[x * bsize + y] = 0.0;
					for (int z = 0; z < nlayer; z++){//2
						hostD[x * bsize + y] += hostA[z * clayer + x] * hostC[z * bsize + y];
					}
				}
			}
			if(!debug) t.lap("CPU serial mmul elapsed time");

			if(debug){
				//print_grid(agrid,ablock);
				//gnn_kernels::printGPU(devA,nlayer,clayer);
				//hipDeviceSynchronize(); printf("<----->\n");
				//gnn_kernels::printGPU(devC,nlayer,bsize);
				//hipDeviceSynchronize(); printf("<----->\n");
				gnn_kernels::printGPU(devB,clayer,bsize);
				hipDeviceSynchronize();
				for(int x = 0; x<clayer * bsize;x++){
					printf("%.4f ", hostD[x]);
					if((x+1)%k==0) printf("\n");
				}
			}else{
				for(int x = 0; x<clayer * bsize;x++){
					if(((hostD[x] - hostB[x]) > 0.001 )){
						printf("Result matrices do not match(%f,%f)!!!\n",hostD[x],hostB[x] );
					}
				}
			}*/
			hipHostFree(hostD);
		}else if (test == MHPROD){
			dim3 dgrid((bsize - 1) / TTILE + 1, (clayer - 1) / TTILE + 1);
			dim3 dblock(TTILE, TTILE);

			if(debug){
				printf("D=");
				gnn_kernels::printGPU(devC,nlayer,bsize);
				hipDeviceSynchronize();
				printf("W=");
				gnn_kernels::printGPU(devA,nlayer,clayer);
				hipDeviceSynchronize();
				printf("A=");
				gnn_kernels::printGPU(devB,clayer-1,bsize);
				hipDeviceSynchronize();
			}
			Time<millis> t;
			t.start();
			gnn_kernels::hmprod_mmul<DATA_T,ACT_F,TTILE><<<dgrid, dblock>>>(
					devC,
					devA,
					devB,
					F,
					nlayer,
					clayer-1,
					bsize
					);
			handleDeviceErrors(hipDeviceSynchronize(),"Error executing hmprod_tmmul kernel");
			t.lap("GPU serial hmprod elapsed time");

			allocHostMem<DATA_T>(&hostD,sizeof(DATA_T) * nlayer * bsize, "Error allocating devC memory");
			safeCpyToHost<DATA_T>(hostA,devA,sizeof(DATA_T)*nlayer*clayer,"Error copying devA to host");
			safeCpyToHost<DATA_T>(hostB,devB,sizeof(DATA_T)*clayer*bsize,"Error copying devB to host");
			safeCpyToHost<DATA_T>(hostC,devC,sizeof(DATA_T)*nlayer*bsize,"Error copying devC to host");

			/*for(int x = 0; x < nlayer; x++){//3
				for (int y = 0; y < bsize; y++){//3
					hostD[x * bsize + y] = hostA[x * (clayer) + clayer - 1];
					for (int z = 0; z < clayer - 1; z++){//2
						hostD[x * bsize + y] += hostA[x * (clayer) + z] * hostB[z * bsize + y];
					}
					hostD[x * bsize + y] = F.f(hostD[x * bsize + y]);
				}
			}*/

			if(debug){
				printf("R=");
				gnn_kernels::printGPU(devC,nlayer,bsize);
				hipDeviceSynchronize();
				for(int x = 0; x<nlayer * bsize;x++){
					printf("%.4f ", hostD[x]);
					if((x+1)%k==0) printf("\n");
				}
			}
		}else if( test == TVECPVEC ){
			dim3 grid((clayer - 1)/TTILE + 1, (nlayer - 1)/TTILE + 1 );
			dim3 block(TTILE,TTILE);

			if(false){
				//print_grid(grid,block);
				printf("W=");
				gnn_kernels::printGPU(devA,nlayer,clayer);
				hipDeviceSynchronize();
				printf("D=");
				gnn_kernels::printGPU(devC,nlayer,bsize);
				hipDeviceSynchronize();
				printf("A=");
				gnn_kernels::printGPU(devB,clayer-1,bsize);
				hipDeviceSynchronize();
				printf("E=tvecpvec(W,D,A,%f,%d)\n",0.3233,bsize);
			}

			Time<millis> t;
			t.start();
			gnn_kernels::tvecpvec<DATA_T,TTILE><<<grid,block>>>(
					devA,
					devC,
					devB,
					nlayer,
					bsize,
					clayer-1,
					0.0231
					);
			handleDeviceErrors(hipDeviceSynchronize(),"Error executing tvecpvec kernel");
			t.lap("GPU serial tvecpvec elapsed time");

			if(debug){
				//printf("R=");
				gnn_kernels::printGPU(devA,nlayer,clayer);
				hipDeviceSynchronize();
				//printf("round(R-E)\n");
			}
		}

		hipFree(devA); hipFree(devB); hipFree(devC);
		hipHostFree(hostA); hipHostFree(hostB); hipHostFree(hostC); hipHostFree(hostD);
		hipDeviceReset();
	}

	template class GNeuralNetwork<float,gnn_actf::Sigmoid>;
	template class GNeuralNetwork<float,gnn_actf::FSigmoid>;
	template class GNeuralNetwork<float,gnn_actf::Arctan>;

	//template class GNeuralNetwork<double,gnn_actf::Sigmoid>;
	//template class GNeuralNetwork<double,gnn_actf::FSigmoid>;
	//template class GNeuralNetwork<double,gnn_actf::Arctan>;
}
